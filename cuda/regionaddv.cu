
#include <hip/hip_runtime.h>
#include <stdint.h>

// add region-based vector to dst:
// dst[i] += LUT[region[i]]
extern "C" __global__ void
regionaddv(float* __restrict__ dstx, float* __restrict__ dsty, float* __restrict__ dstz,
           float* __restrict__ LUTx, float* __restrict__ LUTy, float* __restrict__ LUTz,
           uint16_t* regions, int N) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
    if (i < N) {

        uint16_t r = regions[i];
        dstx[i] += LUTx[r];
        dsty[i] += LUTy[r];
        dstz[i] += LUTz[r];
    }
}


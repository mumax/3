#include "hip/hip_runtime.h"
#include <stdint.h>
//#include "ext_SAFM.h"
#include "float3.h"
#include "stencil.h"
#include "amul.h"

// See exchange.go for more details.
extern "C" __global__ void
addAFMexchange(float* __restrict__ Bx, float* __restrict__ By, float* __restrict__ Bz,
            float* __restrict__ mx, float* __restrict__ my, float* __restrict__ mz,
            float AFMex, 
            int AFMR1, 
            int AFMR2,
	    float tsp,
            float* __restrict__ Ms_, float Ms_mul,
             uint16_t* __restrict__ regions,
            float wx, float wy, float wz, int Nx, int Ny, int Nz, uint16_t PBC) {

    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int iz = blockIdx.z * blockDim.z + threadIdx.z;
 


    if (ix >= Nx || iy >= Ny || iz >= Nz) {
        return;
    }

    int I = idx(ix, iy, iz);
    float3 m0 = make_float3(mx[I], my[I], mz[I]);
    float3 B  = make_float3(Bx[I], By[I], Bz[I]);
    float Ms   = amul(Ms_, Ms_mul, I);
//printf("AFM %f %d %d\n",AFMex,AFMR1,AFMR2); 
    if (AFMex!=0) {

     if (iz==AFMR1)
     {
     int i_;    // neighbor index
     float3 m_; // neighbor mag
 
     i_  = idx(ix, iy, AFMR2);           // clamps or wraps index according to PBC
     m_  = make_float3(mx[i_], my[i_], mz[i_]);  // load m
     m_  = ( is0(m_)? m0: m_ );                  // replace missing non-boundary neighbor
     B += AFMex/Ms/tsp *(m_ - m0);

     }

     if (iz==AFMR2)
     {
     int i_;    // neighbor index
     float3 m_; // neighbor mag
 
     i_  = idx(ix, iy, AFMR1);           // clamps or wraps index according to PBC
     m_  = make_float3(mx[i_], my[i_], mz[i_]);  // load m
     m_  = ( is0(m_)? m0: m_ );                  // replace missing non-boundary neighbor
     B += AFMex/Ms/tsp *(m_ - m0);
     }

    }


    Bx[I] = B.x;
    By[I] = B.y;
    Bz[I] = B.z;

}


#include "hip/hip_runtime.h"
// Original implementation by Mykola Dvornik for mumax2
// Modified for mumax3 by Arne Vansteenkiste, 2013, 2016

#include <stdint.h>
#include "float3.h"
#include "constants.h"
#include "amul.h"

extern "C" __global__ void
addslonczewskitorque2(float* __restrict__ tx, float* __restrict__ ty, float* __restrict__ tz,
                      float* __restrict__ mx, float* __restrict__ my, float* __restrict__ mz,
                      float* __restrict__ Ms_,        float  Ms_mul,
                      float* __restrict__ jz_,        float  jz_mul,
                      float* __restrict__ px_,        float  px_mul,
                      float* __restrict__ py_,        float  py_mul,
                      float* __restrict__ pz_,        float  pz_mul,
                      float* __restrict__ alpha_,     float  alpha_mul,
                      float* __restrict__ pol_,       float  pol_mul,
                      float* __restrict__ lambda_,    float  lambda_mul,
                      float* __restrict__ epsPrime_,  float  epsPrime_mul,
                      float* __restrict__ thickness_, float  thickness_mul,
                      float meshThickness,
                      float freeLayerPosition,
                      int N) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
    if (i < N) {

        float3 m = make_float3(mx[i], my[i], mz[i]);
        float  J = amul(jz_, jz_mul, i);
        float3 p = normalized(vmul(px_, py_, pz_, px_mul, py_mul, pz_mul, i));
        float  Ms           = amul(Ms_, Ms_mul, i);
        float  alpha        = amul(alpha_, alpha_mul, i);
        float  pol          = amul(pol_, pol_mul, i);
        float  lambda       = amul(lambda_, lambda_mul, i);
        float  epsilonPrime = amul(epsPrime_, epsPrime_mul, i);

        float thickness = amul(thickness_, thickness_mul, i);
        if (thickness == 0.0) { // if thickness is not set, use the thickness of the mesh instead
            thickness = meshThickness;
        }
        thickness *= freeLayerPosition; // switch sign if fixedlayer is at the bottom

        if (J == 0.0f || Ms == 0.0f) {
            return;
        }

        float beta    = (HBAR / QE) * (J / (thickness*Ms) );
        float lambda2 = lambda * lambda;
        float epsilon = pol * lambda2 / ((lambda2 + 1.0f) + (lambda2 - 1.0f) * dot(p, m));

        float A = beta * epsilon;
        float B = beta * epsilonPrime;

        float gilb     = 1.0f / (1.0f + alpha * alpha);
        float mxpxmFac = gilb * (A + alpha * B);
        float pxmFac   = gilb * (B - alpha * A);

        float3 pxm      = cross(p, m);
        float3 mxpxm    = cross(m, pxm);

        tx[i] += mxpxmFac * mxpxm.x + pxmFac * pxm.x;
        ty[i] += mxpxmFac * mxpxm.y + pxmFac * pxm.y;
        tz[i] += mxpxmFac * mxpxm.z + pxmFac * pxm.z;
    }
}


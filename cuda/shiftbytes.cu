#include "hip/hip_runtime.h"
#include <stdint.h>
#include "stencil.h"

// shift dst by shx cells (positive or negative) along X-axis.
// new edge value is clampL at left edge or clampR at right edge.
extern "C" __global__ void
shiftbytes(uint16_t* __restrict__  dst, uint16_t* __restrict__  src,
           int Nx,  int Ny,  int Nz, int shx, uint16_t clamp) {

    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int iz = blockIdx.z * blockDim.z + threadIdx.z;

    if(ix < Nx && iy < Ny && iz < Nz) {
        int ix2 = ix-shx;
        uint16_t newval;
        if (ix2 < 0 || ix2 >= Nx) {
            newval = clamp;
        } else {
            newval = src[idx(ix2, iy, iz)];
        }
        dst[idx(ix, iy, iz)] = newval;
    }
}


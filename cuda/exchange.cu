#include "hip/hip_runtime.h"
#include <stdint.h>
#include "exchange.h"
#include "float3.h"
#include "stencil.h"

// See exchange.go for more details.
extern "C" __global__ void
addexchange(float* __restrict__ Bx, float* __restrict__ By, float* __restrict__ Bz,
            float* __restrict__ mx, float* __restrict__ my, float* __restrict__ mz,
            float* __restrict__ aLUT2d, uint16_t* __restrict__ regions,
            float wx, float wy, float wz, int Nx, int Ny, int Nz, uint16_t PBC) {

    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int iz = blockIdx.z * blockDim.z + threadIdx.z;

    if (ix >= Nx || iy >= Ny || iz >= Nz) {
        return;
    }

    // central cell
    int I = idx(ix, iy, iz);
    float3 m0 = make_float3(mx[I], my[I], mz[I]);

    if (is0(m0)) {
        return;
    }

    uint16_t r0 = regions[I];
    float3 B  = make_float3(Bx[I], By[I], Bz[I]);

    int i_;    // neighbor index
    float3 m_; // neighbor mag
    float a__; // inter-cell exchange stiffness

    // left neighbor
    i_  = idx(lclampx(ix-1), iy, iz);           // clamps or wraps index according to PBC
    m_  = make_float3(mx[i_], my[i_], mz[i_]);  // load m
    m_  = ( is0(m_)? m0: m_ );                  // replace missing non-boundary neighbor
    a__ = aLUT2d[symidx(r0, regions[i_])];
    B += wx * a__ *(m_ - m0);

    // right neighbor
    i_  = idx(hclampx(ix+1), iy, iz);
    m_  = make_float3(mx[i_], my[i_], mz[i_]);
    m_  = ( is0(m_)? m0: m_ );
    a__ = aLUT2d[symidx(r0, regions[i_])];
    B += wx * a__ *(m_ - m0);

    // back neighbor
    i_  = idx(ix, lclampy(iy-1), iz);
    m_  = make_float3(mx[i_], my[i_], mz[i_]);
    m_  = ( is0(m_)? m0: m_ );
    a__ = aLUT2d[symidx(r0, regions[i_])];
    B += wy * a__ *(m_ - m0);

    // front neighbor
    i_  = idx(ix, hclampy(iy+1), iz);
    m_  = make_float3(mx[i_], my[i_], mz[i_]);
    m_  = ( is0(m_)? m0: m_ );
    a__ = aLUT2d[symidx(r0, regions[i_])];
    B += wy * a__ *(m_ - m0);

    // only take vertical derivative for 3D sim
    if (Nz != 1) {
        // bottom neighbor
        i_  = idx(ix, iy, lclampz(iz-1));
        m_  = make_float3(mx[i_], my[i_], mz[i_]);
        m_  = ( is0(m_)? m0: m_ );
        a__ = aLUT2d[symidx(r0, regions[i_])];
        B += wz * a__ *(m_ - m0);

        // top neighbor
        i_  = idx(ix, iy, hclampz(iz+1));
        m_  = make_float3(mx[i_], my[i_], mz[i_]);
        m_  = ( is0(m_)? m0: m_ );
        a__ = aLUT2d[symidx(r0, regions[i_])];
        B += wz * a__ *(m_ - m0);
    }

    Bx[I] = B.x;
    By[I] = B.y;
    Bz[I] = B.z;
}


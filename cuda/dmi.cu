#include "hip/hip_runtime.h"
#include <stdint.h>
#include "exchange.h"
#include "float3.h"
#include "stencil.h"

// Exchange + Dzyaloshinskii-Moriya interaction according to
// Bagdanov and Röβler, PRL 87, 3, 2001. eq.8 (out-of-plane symmetry breaking).
// Taking into account proper boundary conditions.
// m: normalized magnetization
// H: effective field in Tesla
// D: dmi strength / Msat, in Tesla*m
// A: Aex/Msat
extern "C" __global__ void
adddmi(float* __restrict__ Hx, float* __restrict__ Hy, float* __restrict__ Hz,
       float* __restrict__ mx, float* __restrict__ my, float* __restrict__ mz,
       float* __restrict__ aLUT2d, float* __restrict__ dLUT2d, uint16_t* __restrict__ regions,
       float cx, float cy, float cz, int Nx, int Ny, int Nz, uint16_t PBC) {

    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int iz = blockIdx.z * blockDim.z + threadIdx.z;

    if (ix >= Nx || iy >= Ny || iz >= Nz) {
        return;
    }

    int I = idx(ix, iy, iz);                      // central cell index
    float3 h = make_float3(Hx[I], Hy[I], Hz[I]);  // add to H
    float3 m0 = make_float3(mx[I], my[I], mz[I]); // central m
    uint16_t r0 = regions[I];
    int i_;                                       // neighbor index

    if(is0(m0)) {
        return;
    }

    // x derivatives (along length)
    {
        float3 m1 = make_float3(0.0f, 0.0f, 0.0f);     // left neighbor
        i_ = idx(lclampx(ix-1), iy, iz);               // load neighbor m if inside grid, keep 0 otherwise
        if (ix-1 >= 0 || PBCx) {
            m1 = make_float3(mx[i_], my[i_], mz[i_]);
        }
        float A1 = aLUT2d[symidx(r0, regions[i_])];    // inter-region Aex
        float D1 = dLUT2d[symidx(r0, regions[i_])];    // inter-region Dex
        if (is0(m1)) {                                 // neighbor missing
            m1.x = m0.x - (-cx * (0.5f*D1/A1) * m0.z); // extrapolate missing m from BC's
            m1.y = m0.y;
            m1.z = m0.z + (-cx * (0.5f*D1/A1) * m0.x);
        }
        h   += (2.0f*A1/(cx*cx)) * (m1 - m0);          // exchange
        h.x += (D1/cx)*(- m1.z);
        h.z -= (D1/cx)*(- m1.x);
    }

    {
        float3 m2 = make_float3(0.0f, 0.0f, 0.0f);     // right neighbor
        i_ = idx(hclampx(ix+1), iy, iz);
        if (ix+1 < Nx || PBCx) {
            m2 = make_float3(mx[i_], my[i_], mz[i_]);
        }
        float A2 = aLUT2d[symidx(r0, regions[i_])];
        float D2 = dLUT2d[symidx(r0, regions[i_])];
        if (is0(m2)) {
            m2.x = m0.x - (cx * (0.5f*D2/A2) * m0.z);
            m2.y = m0.y;
            m2.z = m0.z + (cx * (0.5f*D2/A2) * m0.x);
        }
        h   += (2.0f*A2/(cx*cx)) * (m2 - m0);
        h.x += (D2/cx)*(m2.z);
        h.z -= (D2/cx)*(m2.x);
    }

    // y derivatives (along height)
    {
        float3 m1 = make_float3(0.0f, 0.0f, 0.0f);
        i_ = idx(ix, lclampy(iy-1), iz);
        if (iy-1 >= 0 || PBCy) {
            m1 = make_float3(mx[i_], my[i_], mz[i_]);
        }
        float A1 = aLUT2d[symidx(r0, regions[i_])];
        float D1 = dLUT2d[symidx(r0, regions[i_])];
        if (is0(m1)) {
            m1.x = m0.x;
            m1.y = m0.y - (-cy * (0.5f*D1/A1) * m0.z);
            m1.z = m0.z + (-cy * (0.5f*D1/A1) * m0.y);
        }
        h   += (2.0f*A1/(cy*cy)) * (m1 - m0);
        h.y += (D1/cy)*(- m1.z);
        h.z -= (D1/cy)*(- m1.y);
    }

    {
        float3 m2 = make_float3(0.0f, 0.0f, 0.0f);
        i_ = idx(ix, hclampy(iy+1), iz);
        if  (iy+1 < Ny || PBCy) {
            m2 = make_float3(mx[i_], my[i_], mz[i_]);
        }
        float A2 = aLUT2d[symidx(r0, regions[i_])];
        float D2 = dLUT2d[symidx(r0, regions[i_])];
        if (is0(m2)) {
            m2.x = m0.x;
            m2.y = m0.y - (cy * (0.5f*D2/A2) * m0.z);
            m2.z = m0.z + (cy * (0.5f*D2/A2) * m0.y);
        }
        h   += (2.0f*A2/(cy*cy)) * (m2 - m0);
        h.y += (D2/cy)*(m2.z);
        h.z -= (D2/cy)*(m2.y);
    }

    // only take vertical derivative for 3D sim
    if (Nz != 1) {
        // bottom neighbor
        {
            i_  = idx(ix, iy, lclampz(iz-1));
            float3 m1  = make_float3(mx[i_], my[i_], mz[i_]);
            m1  = ( is0(m1)? m0: m1 );                         // Neumann BC
            float A1 = aLUT2d[symidx(r0, regions[i_])];
            h += (2.0f*A1/(cz*cz)) * (m1 - m0);                // Exchange only
        }

        // top neighbor
        {
            i_  = idx(ix, iy, hclampz(iz+1));
            float3 m2  = make_float3(mx[i_], my[i_], mz[i_]);
            m2  = ( is0(m2)? m0: m2 );
            float A2 = aLUT2d[symidx(r0, regions[i_])];
            h += (2.0f*A2/(cz*cz)) * (m2 - m0);
        }
    }

    // write back, result is H + Hdmi + Hex
    Hx[I] = h.x;
    Hy[I] = h.y;
    Hz[I] = h.z;
}

// Note on boundary conditions.
//
// We need the derivative and laplacian of m in point A, but e.g. C lies out of the boundaries.
// We use the boundary condition in B (derivative of the magnetization) to extrapolate m to point C:
// 	m_C = m_A + (dm/dx)|_B * cellsize
//
// When point C is inside the boundary, we just use its actual value.
//
// Then we can take the central derivative in A:
// 	(dm/dx)|_A = (m_C - m_D) / (2*cellsize)
// And the laplacian:
// 	lapl(m)|_A = (m_C + m_D - 2*m_A) / (cellsize^2)
//
// All these operations should be second order as they involve only central derivatives.
//
//    ------------------------------------------------------------------ *
//   |                                                   |             C |
//   |                                                   |          **   |
//   |                                                   |        ***    |
//   |                                                   |     ***       |
//   |                                                   |   ***         |
//   |                                                   | ***           |
//   |                                                   B               |
//   |                                               *** |               |
//   |                                            ***    |               |
//   |                                         ****      |               |
//   |                                     ****          |               |
//   |                                  ****             |               |
//   |                              ** A                 |               |
//   |                         *****                     |               |
//   |                   ******                          |               |
//   |          *********                                |               |
//   |D ********                                         |               |
//   |                                                   |               |
//   +----------------+----------------+-----------------+---------------+
//  -1              -0.5               0               0.5               1
//                                 x

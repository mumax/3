#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>
#include <math.h>
#include "exchange.h"
#include "float3.h"
#include "stencil.h"

// Returns the topological charge contribution on an elementary triangle ijk
// Order of arguments is important here to preserve the same measure of chirality
// Note: the result is zero if an argument is zero, or when two arguments are the same
__device__ inline float triangleCharge(float3 mi, float3 mj, float3 mk) {
    float numer   = dot(mi, cross(mj, mk));
    float denom   = 1.0f + dot(mi, mj) + dot(mi, mk) + dot(mj, mk);
    return 2.0f * atan2(numer, denom);
}

// Set s to the toplogogical charge density for lattices based on the solid angle 
// subtended by triangle associated with three spins: a,b,c
//
// 	  s = 2 atan[(a . b x c /(1 + a.b + a.c + b.c)] / (dx dy)
//
// After M Boettcher et al, New J Phys 20, 103014 (2018), adapted from
// B. Berg and M. Luescher, Nucl. Phys. B 190, 412 (1981), and implemented by
// Joo-Von Kim.
//
// A unit cell comprises two triangles, but s is a site-dependent quantity so we
// double-count and average over four triangles.
extern "C" __global__ void
settopologicalchargelattice(float* __restrict__ s,
                     float* __restrict__ mx, float* __restrict__ my, float* __restrict__ mz,
                     float icxcy, int Nx, int Ny, int Nz, uint8_t PBC) {

    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int iz = blockIdx.z * blockDim.z + threadIdx.z;

    if (ix >= Nx || iy >= Ny || iz >= Nz) {
        return;
    }

    int i0 = idx(ix, iy, iz);                        // central cell index
    float3 m0 = make_float3(mx[i0], my[i0], mz[i0]); // central cell magnetization

    if(is0(m0)) {
        s[i0] = 0.0f;
        return;
    }

    // indices of the 4 neighbors (counter clockwise)
    int i1 = idx(hclampx(ix+1), iy, iz); // (i+1,j)
    int i2 = idx(ix, hclampy(iy+1), iz); // (i,j+1)
    int i3 = idx(lclampx(ix-1), iy, iz); // (i-1,j)
    int i4 = idx(ix, lclampy(iy-1), iz); // (i,j-1)

    // magnetization of the 4 neighbors
    float3 m1 = make_float3(mx[i1], my[i1], mz[i1]);
    float3 m2 = make_float3(mx[i2], my[i2], mz[i2]);
    float3 m3 = make_float3(mx[i3], my[i3], mz[i3]);
    float3 m4 = make_float3(mx[i4], my[i4], mz[i4]);

    // local topological charge (accumulator)
    float topcharge = 0.0; 

    // charge contribution from the upper right triangle
    // if diagonally opposite neighbor is not zero, use a weight of 1/2 to avoid counting charges twice
    if ((ix+1<Nx || PBCx) && (iy+1<Ny || PBCy)) { 
        int i_ = idx(hclampx(ix+1), hclampy(iy+1), iz); // diagonal opposite neighbor in upper right quadrant
        float3 m_ = make_float3(mx[i_], my[i_], mz[i_]);
        float weight = is0(m_) ? 1 : 0.5;
        topcharge += weight * triangleCharge(m0, m1, m2);
    }

    // upper left
    if ((ix-1>=0 || PBCx) && (iy+1<Ny || PBCy)) { 
        int i_ = idx(lclampx(ix-1), hclampy(iy+1), iz); 
        float3 m_ = make_float3(mx[i_], my[i_], mz[i_]);
        float weight = is0(m_) ? 1 : 0.5;
        topcharge += weight * triangleCharge(m0, m2, m3);
    }

    // bottom left
    if ((ix-1>=0 || PBCx) && (iy-1>=0 || PBCy)) { 
        int i_ = idx(lclampx(ix-1), lclampy(iy-1), iz); 
        float3 m_ = make_float3(mx[i_], my[i_], mz[i_]);
        float weight = is0(m_) ? 1 : 0.5;
        topcharge += weight * triangleCharge(m0, m3, m4);
    }

    // bottom right
    if ((ix+1<Nx || PBCx) && (iy-1>=0 || PBCy)) { 
        int i_ = idx(hclampx(ix+1), lclampy(iy-1), iz); 
        float3 m_ = make_float3(mx[i_], my[i_], mz[i_]);
        float weight = is0(m_) ? 1 : 0.5;
        topcharge += weight * triangleCharge(m0, m4, m1);
    }

    s[i0] = icxcy * topcharge;
}

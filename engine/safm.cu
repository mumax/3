#include "hip/hip_runtime.h"
#include <stdint.h>
#include "ext_SAFM.h"
#include "float3.h"
#include "stencil.h"

// See exchange.go for more details.
extern "C" __global__ void
addAFMexchange(float* __restrict__ Bx, float* __restrict__ By, float* __restrict__ Bz,
            float* __restrict__ mx, float* __restrict__ my, float* __restrict__ mz,
            float AFMex, 
            int AFMR1, 
            int AFMR2, 
            uint8_t* __restrict__ regions,
            float wx, float wy, float wz, int Nx, int Ny, int Nz, uint8_t PBC) {

    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int iz = blockIdx.z * blockDim.z + threadIdx.z;
 


    if (ix >= Nx || iy >= Ny || iz >= Nz) {
        return;
    }

    int I = idx(ix, iy, iz);
    float3 m0 = make_float3(mx[I], my[I], mz[I]);
    float3 B  = make_float3(Bx[I], By[I], Bz[I]);
//printf("AFM %f %d %d\n",AFMex,AFMR1,AFMR2); 
    if (AFMex!=0) {

     if (iz==AFMR1)
     {
     int i_;    // neighbor index
     float3 m_; // neighbor mag
 
     i_  = idx(ix, iy, AFMR2);           // clamps or wraps index according to PBC
     m_  = make_float3(mx[i_], my[i_], mz[i_]);  // load m
     m_  = ( is0(m_)? m0: m_ );                  // replace missing non-boundary neighbor
     B += AFMex *(m_ - m0);

     }

     if (iz==AFMR2)
     {
     int i_;    // neighbor index
     float3 m_; // neighbor mag
 
     i_  = idx(ix, iy, AFMR1);           // clamps or wraps index according to PBC
     m_  = make_float3(mx[i_], my[i_], mz[i_]);  // load m
     m_  = ( is0(m_)? m0: m_ );                  // replace missing non-boundary neighbor
     B += AFMex *(m_ - m0);
     }

    }


    Bx[I] = B.x;
    By[I] = B.y;
    Bz[I] = B.z;

}


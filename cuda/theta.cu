#include "hip/hip_runtime.h"
#include "stencil.h"

extern "C" __global__ void
setTheta(float* __restrict__ theta, float* __restrict__ mz, int Nx, int Ny, int Nz) {

    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int iz = blockIdx.z * blockDim.z + threadIdx.z;

    if (ix >= Nx || iy >= Ny || iz >= Nz)
    {
        return;
    }

    int I = idx(ix, iy, iz);                      // central cell index
    theta[I] = acosf(mz[I]);
}